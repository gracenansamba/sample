//********************************************************************************************************

#include <hip/hip_runtime.h>
#include <cstdio>// a simple matrix matrix multiplication in CUDA
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>

using namespace std;

//#ifdef DOLOG
//#define LOG(msg) std::cerr<<msg<<std::endl
#define LOG(msg) fprintf(stderr, msg "\n");
//#else
//#define LOG(msg)
//#endif

// host code for validating last cuda operation (not kernel launch)
//using namespace std;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// CPU matrix multiplication: code(the normal way to mulitiply matrices)
__global__
void  multiply (int width,int height ,int firstCol ,int** A,int** B,int** C,int** A_O,int** B_O, int** C_O)
{

int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
int y_idx = threadIdx.y + blockIdx.y * blockDim.y;


if (x_idx < width && y_idx <height ) {
//int temp = 0;

  for (int i = 0; i <3; i++)
    {
      //printf ("i= %d", i);
      //printf ("\n");
      for (int j = 0; j <3; j++)
	{
 	//printf ("j= %d", j);
 	//printf ("\n");
	int temp = 0;
	//printf ("temp= %d", temp);
 	//printf ("\n");
 	for (int k = 0; k < firstCol; k++)
  	 {
     		//printf ("k= %d", k);
    		// printf ("\n");
     		temp += A[i][k] * B[k][j]; //C is fixed while A is rowWise and B is columnWise
     		C_O[i][j] =temp ;
     		//printf ("Cij= %d", temp);
    		//printf ("\n");
 	}
    }
  }
}

}
//**************************************************************************************
__global__ void display (int width, int height,int** A,int** B,int** C ){

	for (int i = 0; i <3; i++)
	{
		for (int j = 0; j < 3; j++)
		{
 		//printf ("%d\t", C[i][j]);
		}
	printf ("\n");
   }
}
//***********************************************************************************
int main ()
{

 int width, height;
//vector<vector<int> > A { {1, 2, 3}, {4, 5, 6}, {7, 8, 9} };
//vector<vector<int> > B { {1, 2, 3}, {4, 5, 6}, {7, 8, 9} };
//vector<vector<int> > C { {0, 0, 0}, {0, 0, 0}, {0, 0, 0} };

int A[][3]= { {1, 2, 3}, {4, 5, 6}, {7, 8, 9} };
int B[][3]= { {1, 2, 3}, {4, 5, 6}, {7, 8, 9} };
int C[][3]= { {0, 0, 0}, {0, 0, 0}, {0, 0, 0} };

//**********************************************************************************
//Memory allocaction

   width=3;
    height=3;
int firstCol= 3;
int **d_A, **d_B, **d_C; //allocate memory on device
//copy matrix to GPU
gpuErrchk(hipMalloc((void**)&d_A, (width)*(height)*sizeof(int)));
gpuErrchk(hipMalloc((void**)&d_B, (width)*(height)*sizeof(int)));
gpuErrchk(hipMalloc((void**)&d_C, (width)*(height)*sizeof(int)));

//same
int **A_O,**B_O,**C_O;
gpuErrchk(hipMalloc((void**)&A_O,(width)*(height)*sizeof(int)));
gpuErrchk(hipMalloc((void**)&B_O,(width)*(height)*sizeof(int)));
gpuErrchk(hipMalloc((void**)&C_O,(width)*(height)*sizeof(int)));


gpuErrchk(hipMemcpy(d_A, &A[0][0],(width)*(height)*sizeof(int) , hipMemcpyHostToDevice));
gpuErrchk(hipMemcpy(d_B, &B[0][0],(width)*(height)*sizeof(int) , hipMemcpyHostToDevice));
gpuErrchk(hipMemcpy(d_C, &C[0][0],(width)*(height)*sizeof(int) , hipMemcpyHostToDevice));
//printf ("matrix A= %d", A);
//printf ("matrix B= %d", B);
//call kernel
dim3 tpb(16,16);
dim3 bpg((width+tpb.x-1)/tpb.x, (height+tpb.y-1)/tpb.y);

multiply<<<bpg,tpb>>>(width, height , firstCol ,d_A,d_B,d_C, A_O, B_O, C_O);
display<<<bpg,tpb>>>(width,height,d_A,d_B,d_C);

//copy matrix back to CPU

//gpuErrchk(cudaMemcpy(&A[0][0], d_A, (width)*(height)*sizeof(int), cudaMemcpyDeviceToHost));
//gpuErrchk(cudaMemcpy(&B[0][0], d_B, (width)*(height)*sizeof(int), cudaMemcpyDeviceToHost));
gpuErrchk(hipMemcpy(&C[0][0], (void**)C_O, (width)*(height)*sizeof(int), hipMemcpyDeviceToHost));

hipFree(d_A);hipFree(d_B);hipFree(d_C);hipFree(A_O);hipFree(B_O);hipFree(C_O);

return 0;
}