#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#define blockSize 1

using namespace std;

__global__
void dots(float *A, float *B, float *C, int M, int N, int K) 
{
	int I = blockIdx.x * blockDim.x + threadIdx.x;
	int J = blockIdx.y * blockDim.y + threadIdx.y;

	if (I>=M || J >=N)
		printf("threads are more than size");
	
	else 
		{
			int k; //where do  i set the k from? 
			printf("A : %d, B : %d, C : %d, M : %d, N : %d, K : %d\n", A,B,C,M,N,K);
			
			float temp = 0;
			for (k=0; k<K; k++){
				temp += A[I*K+k] * B[k*N +J];
				C[I*N+J] = temp;
				printf ("C = %d  " ,C[I*N+J]);
			}
			//printf("CIJ = %d\n", C[I*N+J]);
			printf("AIk = %d\n", A[I*K+k]);
			printf("BkJ = %d\n", B[k*N+J]);
			
		}
}
int main()
{
	int M=3; int N=3; int K =1;
	int A[M][K]= {1,2,3}; //can i do a dot like a matrix?
	int B[K][N]= {1,2,3};
	int C[M][N] = {0,0,0};
/*
	is this how i would do a normal dot
	int A[M]= {1,2,3};
	int B[N]= {1,2,3};
	int C[K] = {0,0,0};
	
*/
	
//Allocate memory for device
	float *dA, *dB, *dC;
	hipMalloc(&dA, M*sizeof(float));
	hipMalloc(&dB, N*sizeof(float));
	hipMalloc(&dC, K*sizeof(float));

//Copy memory to deveice

	hipMemcpy(&dA, A, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&dB, B, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&dC, C, sizeof(float), hipMemcpyDeviceToHost);

//call kernel
	
	dots<<<1,1>>>(dA,dB,dC,M,N,K);

//Copy memory back to host
	hipMemcpy(C,&dC, sizeof(float), hipMemcpyHostToDevice);
	
	hipFree (dA);  hipFree (dB);  hipFree (dC);
	return 0;
}
