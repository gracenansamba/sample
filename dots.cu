#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#define blockSize 1

using namespace std;

__global__
void dots(float *A, float *B, float *C, int M, int N, int K) 

{	
	 printf("A : %.1f %.1f %.1f", A[0], A[1] ,A[2]);
       	printf("B : %.1f %.1f %.1f", B[0] ,B[1], B[2]);
	printf ("C: %.1f  %.1f %.1f", C[0] , C[1], C[2]);
 

	int I = blockIdx.x * blockDim.x + threadIdx.x;
	int J = blockIdx.y * blockDim.y + threadIdx.y;

	if (I>=M || J >=N)
		printf("threads are more than size");

	
	else 
		{
			int k; //where do  i set the k from? 
			//printf("A : %d", A[0]);
			
			float temp = 0;
			for (k=0; k<3; k++){
				temp += A[I*K+k] * B[k*N +J];
				C[I*N+J] = temp;
				printf ("C = %d  " ,C[I*N+J]);
			}
			//printf("CIJ = %d\n", C[I*N+J]);
			printf("AIk = %d\n", A[I*K+k]);
			printf("BkJ = %d\n", B[k*N+J]);
			
		}
}
int main()
{

	int M=3; int N=3; int K =3;
/*
	float A[M][K]= {1,2,3}; //can i do a dot like a matrix?
	float B[K][N]= {1,2,3};
	float C[M][N] = {0,0,0};
*/
	//is this how i would do a normal dot
	float  A[M]= {1,2,3};//printf("A : % d  %d %d ", A[0], A[1], A[2]);
		//float *A = Aa; printf("A : % d  %d %d ", A[0], A[1], A[2]);
	float B[N]= {1,2,3};
	float C[3] = {0,0,0};
	

	
//Allocate memory for device
	float *dA, *dB, *dC;
	hipMalloc(&dA, M*sizeof(float));
	hipMalloc(&dB, N*sizeof(float));
	hipMalloc(&dC, sizeof(float));

//Copy memory to deveice

	hipMemcpy(dA, &A[0], M* sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dB, &B[0], N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dC, &C[0], sizeof(float),hipMemcpyHostToDevice);

//call kernel
	
	dots<<<1,1>>>(dA,dB,dC,M,N,K);

//Copy memory back to host
	hipMemcpy(C,&dC, sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree (dA);  hipFree (dB);  hipFree (dC);
	return 0;
}

