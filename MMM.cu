#include "hip/hip_runtime.h"
#include<cmath> 
#include<cstdio> 

__global__ 
void multiply(int*A,int*B,int*C, int M, int N, int K, int n) 
	{ 
	int i=blockIdx.y* blockDim.y + threadIdx.y; printf(" i= %d \n", i);
				//printf(" Tx= %d \n", threadIdx.y);
	int j=blockIdx.x *blockDim.x + threadIdx.x; printf(" j= %d \n", j);
				//printf(" Ty= %d \n", threadIdx.x);
		//for(int i=0;i<n;i++){    printf("%d\n",A[i]);} printf("\n", " ");
		//for(int j=0;j<n;j++){  printf("%d\n",B[j]);} 
	int temp=0;
	if( (i<(n)) && (j<(n))) { 
		
		for( int k=0;k<n;k++){printf("\n", " "); printf(" k= %d \n", k);
			 temp += A[(i)*N+k]*B[k*N +j];
				printf(" aIndex= %d \n", i*N+k);
				printf(" a= %d \n", A[(i)*K+k]);

				printf(" bIndex= %d \n", k*N+j);
				printf(" b= %d \n", B[k*N+(j)]);
	
		} C[(i)*N+(j)] =temp;
		for (i = 0; i < n; i++){  
			printf("[%d] =%d\n",i, C[i*N+j]); 
		} 
	} 
}
int main()
{ 
int M=3;int N=3; int K=3; int n=9; 

	int A[M][K]= {{1,2,3}, {4,5,6}, {7,8,9}}; 
	int B[K][N]= {{1,2,3}, {4,5,6}, {7,8,9}};  
	int C[M][N]= {{0,0,0},{0,0,0},{0,0,0}}; 
	
int* d_A;int* d_B;int* d_C; 
//int* A;int* B;int* C;  

//allocating space for variables on device  

hipMalloc(&d_A,n* sizeof(int));//let memory store that m*n space for you of size ints  
hipMalloc(&d_B,n* sizeof(int)); 
hipMalloc(&d_C,n* sizeof(int)); 

//copy Aand B FROM HOST TO DEVICE  

hipMemcpy(d_A, &A[0],n* sizeof(int) , hipMemcpyHostToDevice); 
hipMemcpy(d_B, &B[0],n *sizeof(int) , hipMemcpyHostToDevice); 
hipMemcpy(d_C, &C[0],n*sizeof(int) , hipMemcpyHostToDevice); 


multiply<<<1,1>>>(d_A,d_B,d_C,M,N,K,n); 
//COPY RESULT BACK TO HOST 
hipMemcpy(&C[0], d_C,n* sizeof(int), hipMemcpyDeviceToHost); 


//printf("%d", C[0]);

hipFree(A);//TO FREE MEMORY 

hipFree(B); 
hipFree(C); 
}